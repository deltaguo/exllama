#include "hip/hip_runtime.h"
#include "q4_matmul.cuh"
#include "column_remap.cuh"
#include "../util.cuh"
#include "../matrix.cuh"
#include "../cuda_compat.cuh"
#include "../cuda_buffers.cuh"
#if defined(USE_ROCM)
#include <rocprim/rocprim.hpp>
#include <hip/hip_ext.h>
#include "rocwmma/rocwmma.hpp"
#include "../hip_compat.cuh"
const int THREADS_X = 64; // Block size and thread count along columns in w and out
#else
const int THREADS_X = 32; // Block size and thread count along columns in w and out
#endif
const int THREADS_Y = 1; // Block size and thread count along rows in x and out

#if defined(USE_SMEM)
const int GROUP_STEP = 32; // Assumed group size when block_size_z % groupsize != 0
#endif

#if defined(USE_ROCM)
template <typename Y, typename X>
__host__ __device__ constexpr Y bit_cast(const X &x)
{
    union AsType
    {
        X x;
        Y y;
    };
    return AsType{x}.y;
}

// transpose fp16 2x2
__device__ void transpose_fp16_2x2_register(const half2 &x0, const half2 &x1, half2 &y0, half2 &y1)
{
    constexpr int32_t m0 = 0x05040100;
    constexpr int32_t m1 = 0x07060302;

    // ex: v_perm_b32(0x 11 22 33 44, 0x 55 66 77 88, 0x 05 04 01 00) -> 0x33447788
    //                   -- -- -- --     -- -- -- --      -  -  -  -
    //             index  7  6  5  4      3  2  1  0     33 44 77 88
    // index is reversed because of little endianness (least significant bits first)
    y0 = bit_cast<half2>(__builtin_amdgcn_perm(bit_cast<int32_t>(x1), bit_cast<int32_t>(x0), m0));
    y1 = bit_cast<half2>(__builtin_amdgcn_perm(bit_cast<int32_t>(x1), bit_cast<int32_t>(x0), m1));
}
__device__ __forceinline__ void dot_product_8_wmma(
    half &acc0,
    half &acc1,
    half &acc2,
    half &acc3,
    MatrixView_half &h_,
    const int h_row,
    const int h_row_end,
    const int h_column, // divisible by 8
    MatrixView_q4_column &v_,
    const int v_row, // divisible by 8
    const int v_column,
    const half v_scale,
    const uint32_t v_zero, // + 1 (!!)
    const int count)
{
    using int16_tx2 = __attribute__((__vector_size__(2 * sizeof(int16_t)))) int16_t;
    using halfx4 = __attribute__((__vector_size__(4 * sizeof(__fp16)))) __fp16;
    using floatx4 = __attribute__((__vector_size__(4 * sizeof(float)))) float;

#define INT_16X2(pointer) ((bit_cast<int16_tx2 *>((void *)&(pointer)))[0])
#define INT_32(pointer) ((bit_cast<int32_t *>((void *)&(pointer)))[0])
#define HALF(pointer) ((bit_cast<half *>((void *)&(pointer)))[0])
#define HALF2(pointer) ((bit_cast<half2 *>((void *)&(pointer)))[0])
#define HALFX4(pointer) ((bit_cast<halfx4 *>((void *)&(pointer)))[0])
#define FLOATX4(pointer) ((bit_cast<floatx4 *>((void *)&(pointer)))[0])

    const int K_tile = 32;
    const half *__restrict__ h_ptr = h_.item_ptr(h_row, h_column);
    uint32_t *__restrict__ v_ptr = (uint32_t *)v_.item_uint32_ptr(v_row, v_column);
    short v_zero_short = (short)v_zero;

    int16_t v_zero_2_add16[2];
    v_zero_2_add16[0] = 16 - v_zero_short;
    v_zero_2_add16[1] = 16 - v_zero_short;

    half2 val_1040_half2;
    val_1040_half2 = __half2half2(__float2half(1040));

    half fragA[K_tile / 4][4];
    half fragB[K_tile / 4][4];
    float fragAcc[4] = {(0.0f)};

    int16_t v_read_B_q[K_tile / 8][2];
    constexpr int sh_A_row_size = 128;
    __shared__ half sh_A[sh_A_row_size * 4];

#pragma unroll
    for (int i = 0; i < K_tile / 8; i++)
    {
        HALF2(v_read_B_q[i][0]) = HALF2(*v_ptr);
        v_ptr += v_.width;
    }

// 转化数据并放入fragB中
#pragma unroll
    for (int k_index = 0; k_index < K_tile / 4; k_index++)
    {
        fragB[k_index][0] = __short2half_rn((short)((v_read_B_q[k_index / 2][k_index % 2] >> (0)) & 0x0f) - v_zero_short);
        fragB[k_index][1] = __short2half_rn((short)((v_read_B_q[k_index / 2][k_index % 2] >> (4)) & 0x0f) - v_zero_short);
        fragB[k_index][2] = __short2half_rn((short)((v_read_B_q[k_index / 2][k_index % 2] >> (8)) & 0x0f) - v_zero_short);
        fragB[k_index][3] = __short2half_rn((short)((v_read_B_q[k_index / 2][k_index % 2] >> (12)) & 0x0f) - v_zero_short);
    }

    int j = 0;

    for (; j < count - K_tile / 8; j = j + K_tile / 8)
    {
#pragma unroll
        for (int i = 0; i < K_tile / 8; i++)
        {
            HALF2(v_read_B_q[i][0]) = HALF2(*v_ptr);
            v_ptr += v_.width;
        }

        if ((j * 8) % sh_A_row_size == 0)
        {
            for (int i = 0; i < h_row_end - h_row; ++i)
            {
                HALF2(sh_A[2 * threadIdx.x + i * sh_A_row_size]) = HALF2(*(h_ptr + h_.width * i + (j * 8) + 2 * threadIdx.x));
            }
        }

        __syncthreads();

#pragma unroll
        for (int k_index = 0; k_index < K_tile / 4; k_index++)
        {
            HALFX4(fragA[k_index][0]) = HALFX4(*(sh_A + (j * 8) % 128 / K_tile * K_tile + k_index * 4 + (threadIdx.x % 4) * sh_A_row_size));
        }

#pragma unroll
        for (int k_index = 0; k_index < K_tile / 8; k_index++)
        {
            int16_t tmp0[2];
            INT_32(tmp0[0]) = (((INT_32(v_read_B_q[k_index][0]) >> 0) & 0x000f000f) + INT_32(v_zero_2_add16[0])) | (0x64006400);
            HALF2(tmp0[0]) = __hsub2(HALF2(tmp0[0]), val_1040_half2);

            int16_t tmp1[2];
            INT_32(tmp1[0]) = (((INT_32(v_read_B_q[k_index][0]) >> 4) & 0x000f000f) + INT_32(v_zero_2_add16[0])) | (0x64006400);
            HALF2(tmp1[0]) = __hsub2(HALF2(tmp1[0]), val_1040_half2);

            int16_t tmp2[2];
            INT_32(tmp2[0]) = (((INT_32(v_read_B_q[k_index][0]) >> 8) & 0x000f000f) + INT_32(v_zero_2_add16[0])) | (0x64006400);
            HALF2(tmp2[0]) = __hsub2(HALF2(tmp2[0]), val_1040_half2);

            int16_t tmp3[2];
            INT_32(tmp3[0]) = (((INT_32(v_read_B_q[k_index][0]) >> 12) & 0x000f000f) + INT_32(v_zero_2_add16[0])) | (0x64006400);
            HALF2(tmp3[0]) = __hsub2(HALF2(tmp3[0]), val_1040_half2);

            FLOATX4(fragAcc[0]) = __builtin_amdgcn_mfma_f32_4x4x4f16(HALFX4(fragA[2 * k_index][0]), HALFX4(fragB[2 * k_index][0]), FLOATX4(fragAcc[0]), 0, 0, 0);
            FLOATX4(fragAcc[0]) = __builtin_amdgcn_mfma_f32_4x4x4f16(HALFX4(fragA[2 * k_index + 1][0]), HALFX4(fragB[2 * k_index + 1][0]), FLOATX4(fragAcc[0]), 0, 0, 0);

            transpose_fp16_2x2_register(HALF2(tmp0[0]), HALF2(tmp1[0]), HALF2(fragB[2 * k_index][0]), HALF2(fragB[2 * k_index + 1][0]));
            transpose_fp16_2x2_register(HALF2(tmp2[0]), HALF2(tmp3[0]), HALF2(fragB[2 * k_index][2]), HALF2(fragB[2 * k_index + 1][2]));
        }
    }

    if ((j * 8) % sh_A_row_size == 0 && threadIdx.x < K_tile)
    {
        for (int i = 0; i < h_row_end - h_row; ++i)
        {
            sh_A[threadIdx.x + i * sh_A_row_size] = *(h_ptr + h_.width * i + (j * 8) + threadIdx.x);
        }
    }

    __syncthreads();

#pragma unroll
    for (int k_index = 0; k_index < K_tile / 4; k_index++)
    {
        HALFX4(fragA[k_index][0]) = HALFX4(*(sh_A + (j * 8) % 128 / K_tile * K_tile + k_index * 4 + (threadIdx.x % 4) * sh_A_row_size));
    }

#pragma unroll
    for (int k_index = 0; k_index < K_tile / 4; k_index++)
    {
        FLOATX4(fragAcc[0]) = __builtin_amdgcn_mfma_f32_4x4x4f16(HALFX4(fragA[k_index][0]), HALFX4(fragB[k_index][0]), FLOATX4(fragAcc[0]), 0, 0, 0);
    }

    switch (h_row_end - h_row)
    {
    case 1:
        acc0 = __hadd(__hmul(__float2half(fragAcc[0]), v_scale), acc0);
        break;
    case 2:
        acc0 = __hadd(__hmul(__float2half(fragAcc[0]), v_scale), acc0);
        acc1 = __hadd(__hmul(__float2half(fragAcc[1]), v_scale), acc1);
        break;
    case 3:
        acc0 = __hadd(__hmul(__float2half(fragAcc[0]), v_scale), acc0);
        acc1 = __hadd(__hmul(__float2half(fragAcc[1]), v_scale), acc1);
        acc2 = __hadd(__hmul(__float2half(fragAcc[2]), v_scale), acc2);
        break;
    case 4:
        acc0 = __hadd(__hmul(__float2half(fragAcc[0]), v_scale), acc0);
        acc1 = __hadd(__hmul(__float2half(fragAcc[1]), v_scale), acc1);
        acc2 = __hadd(__hmul(__float2half(fragAcc[2]), v_scale), acc2);
        acc3 = __hadd(__hmul(__float2half(fragAcc[3]), v_scale), acc3);
        break;
    default:
        break;
    }
}
#endif

typedef void (*fp_q4_matmul_kernel)(
    const half *,
    const uint32_t *,
    half *,
    const half *,
    const uint32_t *,
    const int,
    const int,
    const int,
    const int,
#ifdef WMMA_KERNEL
    const int,
#endif 
    const int,
    const uint32_t *,
    bool);

template <bool use_half2, bool use_groupsize, bool use_x_map>
__global__ void q4_matmul_kernel(
    const half *__restrict__ x,
    const uint32_t *__restrict__ w,
    half *__restrict__ out,
    const half *__restrict__ w_scales,
    const uint32_t *__restrict__ w_zeros,
    const int height,
    const int dim,
    const int width,
    const int groupsize,
#ifdef WMMA_KERNEL
    const int block_size_y,
#endif
    const int block_size_z,
    const uint32_t *__restrict__ x_map,
    bool no_zero)
{
#ifndef WMMA_KERNEL
#if defined(USE_SMEM)

    extern __shared__ half2 x_cache[];
    half *x_cache_h = (half *)x_cache;

#endif

    // Start of block

    int x_column = block_size_z * blockIdx.z;
    int x_column_end = min(dim, block_size_z * (blockIdx.z + 1));

    int w_column = THREADS_X * blockIdx.x + threadIdx.x; // assume width of weight matrix divisible by THREADS_X
    int x_row = THREADS_Y * blockIdx.y + threadIdx.y;

    int iterations = (x_column_end - x_column) / 8;

    // Views

    MatrixView_half x_(x, height, dim);
    MatrixView_half w_scales_(w_scales, dim / groupsize, width);
    MatrixView_q4_row w_zeros_(w_zeros, dim / groupsize, width);
    MatrixView_q4_column w_(w, dim, width);
    MatrixView_half_rw out_(out, height, width);

    // Zero output

    if (!no_zero && blockIdx.z == 0 && (threadIdx.x & 1) == 0)
    {
        *((uint32_t *)out_.item_ptr(x_row, w_column)) = 0;
    }
    __syncthreads();

    // Loop over part of x row (and w column)

    half2 acc = {};
    half acc_h = {};

    if constexpr (use_groupsize)
    {
        // For quant matrices where groupsize divides BLOCK_SIZE_Z we always start on a group boundary, so this
        // could be slightly faster

        for (int k = x_column, group = x_column / groupsize; k < x_column + iterations * 8; group++, k += groupsize)
        {
#if defined(USE_SMEM)

            for (int i = threadIdx.x; i < groupsize; i += THREADS_X)
            {
                if constexpr (use_x_map)
                    x_cache_h[i] = *x_.item_ptr(x_row, x_map[k + i]);
                else
                    x_cache_h[i] = *x_.item_ptr(x_row, k + i);
            }
            __syncthreads();

            if constexpr (use_half2)
            {
                half2 w_scale = w_scales_.item_half2half2(group, w_column);
                uint32_t w_zero = w_zeros_.item(group, w_column) + 1;
                acc = dot_product_8(acc, x_cache, w_, k, w_column, w_scale, w_zero, groupsize / 8);
            }
            else
            {
                half w_scale = w_scales_.item(group, w_column);
                uint32_t w_zero = w_zeros_.item(group, w_column) + 1;
                acc_h = dot_product_8_h(acc_h, x_cache_h, w_, k, w_column, w_scale, w_zero, groupsize / 8);
            }
            __syncthreads();

#else

            if constexpr (use_half2)
            {
                half2 w_scale = w_scales_.item_half2half2(group, w_column);
                uint32_t w_zero = w_zeros_.item(group, w_column) + 1;

                if constexpr (use_x_map)
                    acc = dot_product_8_x_map(acc, x_, x_row, k, w_, k, w_column, w_scale, w_zero, groupsize / 8, x_map);
                else
                    acc = dot_product_8(acc, (const half2 *)x_.item_ptr(x_row, k), w_, k, w_column, w_scale, w_zero, groupsize / 8);
            }
            else
            {
                half w_scale = w_scales_.item(group, w_column);
                uint32_t w_zero = w_zeros_.item(group, w_column) + 1;

                if constexpr (use_x_map)
                    acc_h = dot_product_8_x_map_h(acc_h, x_, x_row, k, w_, k, w_column, w_scale, w_zero, groupsize / 8, x_map);
                else
                    acc_h = dot_product_8_h(acc_h, x_.item_ptr(x_row, k), w_, k, w_column, w_scale, w_zero, groupsize / 8);
            }

#endif
        }
    }
    else
    {
        // Otherwise assume groupsize is a multiple of GROUP_STEP, do GROUP_STEP columns per iteration and trust the cache

#if defined(USE_SMEM)

        for (int k = x_column; k < x_column + iterations * 8; k += GROUP_STEP)
        {
            for (int i = threadIdx.x; i < GROUP_STEP; i += THREADS_X)
            {
                if constexpr (use_x_map)
                    x_cache_h[i] = *x_.item_ptr(x_row, x_map[k + i]);
                else
                    x_cache_h[i] = *x_.item_ptr(x_row, k + i);
            }
            __syncthreads();

            if constexpr (use_half2)
            {
                int group = k / groupsize;
                half2 w_scale = w_scales_.item_half2half2(group, w_column);
                uint32_t w_zero = w_zeros_.item(group, w_column) + 1;
                acc = dot_product_8(acc, x_cache, w_, k, w_column, w_scale, w_zero, GROUP_STEP / 8);
            }
            else
            {
                int group = k / groupsize;
                half w_scale = w_scales_.item(group, w_column);
                uint32_t w_zero = w_zeros_.item(group, w_column) + 1;
                acc_h = dot_product_8_h(acc_h, x_cache_h, w_, k, w_column, w_scale, w_zero, GROUP_STEP / 8);
            }
            __syncthreads();
        }

#else

        for (int k = x_column; k < x_column + iterations * 8; k += 8)
        {
            if constexpr (use_half2)
            {
                int group = k / groupsize;
                half2 w_scale = w_scales_.item_half2half2(group, w_column);
                uint32_t w_zero = w_zeros_.item(group, w_column) + 1;

                if constexpr (use_x_map)
                    acc = dot_product_8_x_map(acc, x_, x_row, k, w_, k, w_column, w_scale, w_zero, 1, x_map);
                else
                    acc = dot_product_8(acc, (const half2 *)x_.item_ptr(x_row, k), w_, k, w_column, w_scale, w_zero, 1);
            }
            else
            {
                int group = k / groupsize;
                half w_scale = w_scales_.item(group, w_column);
                uint32_t w_zero = w_zeros_.item(group, w_column) + 1;

                if constexpr (use_x_map)
                    acc_h = dot_product_8_x_map_h(acc_h, x_, x_row, k, w_, k, w_column, w_scale, w_zero, 1, x_map);
                else
                    acc_h = dot_product_8_h(acc_h, x_.item_ptr(x_row, k), w_, k, w_column, w_scale, w_zero, 1);
            }
        }

#endif
    }

    // Add to block result

    if constexpr (use_half2)
    {
#if defined(USE_SMEM)
        half result = __hadd(__low2half(acc), __low2high(acc));
#else
        half result = __hadd(acc.x, acc.y);
#endif
        atomicAdd(out_.item_ptr(x_row, w_column), result);
    }
    else
    {
        atomicAdd(out_.item_ptr(x_row, w_column), acc_h);
    }
#else
    int x_column = block_size_z * blockIdx.z;
    int x_column_end = min(dim, block_size_z * (blockIdx.z + 1));
    int w_column = THREADS_X * blockIdx.x + threadIdx.x; // assume width of weight matrix divisible by THREADS_X (32)
    int x_row = block_size_y * blockIdx.y;               // 0
    int x_row_end = min(height, block_size_y * (blockIdx.y + 1));

    int iterations = (x_column_end - x_column) / 8;

    // Views
    MatrixView_half x_(x, height, dim);
    MatrixView_half w_scales_(w_scales, dim / groupsize, width);
    MatrixView_q4_row w_zeros_(w_zeros, dim / groupsize, width);
    MatrixView_q4_column w_(w, dim, width);
    MatrixView_half_rw out_(out, height, width);

    // Zero output

    if (!no_zero && blockIdx.z == 0 && (threadIdx.x & 1) == 0)
    {
        for (int i = 0; i < (x_row_end - x_row); ++i)
        {
            *((uint32_t *)out_.item_ptr(x_row + i, w_column)) = 0;
        }
    }
    __syncthreads();

    // Loop over part of x row (and w column)
    // half acc[4] = {__float2half(0), __float2half(0), __float2half(0), __float2half(0)};

    half acc0 = {__float2half(0)};
    half acc1 = {__float2half(0)};
    half acc2 = {__float2half(0)};
    half acc3 = {__float2half(0)};
    if constexpr (use_groupsize)
    {

        for (int k = x_column, group = x_column / groupsize; k < x_column + iterations * 8; group++, k += groupsize)
        {
            {
                half w_scale = w_scales_.item(group, w_column);
                uint32_t w_zero = w_zeros_.item(group, w_column) + 1;

                dot_product_8_wmma(acc0, acc1, acc2, acc3, x_, x_row, x_row_end, k, w_, k, w_column, w_scale, w_zero, min(block_size_z / 8, groupsize / 8));
            }
        }
    }

    __shared__ half sh_result[4 * THREADS_X];
    
    if (x_row_end - x_row > 0)
    {
        sh_result[threadIdx.x] = acc0;
        __syncthreads();
        if (threadIdx.x % 2 == 0)
            atomicAdd((half2 *)out_.item_ptr(x_row, w_column), HALF2(sh_result[threadIdx.x]));
        if (x_row_end - x_row > 1)
        {
            sh_result[threadIdx.x + THREADS_X] = acc1;
            __syncthreads();
            if (threadIdx.x % 2 == 0)
                atomicAdd((half2 *)out_.item_ptr(x_row + 1, w_column), HALF2(sh_result[THREADS_X + threadIdx.x]));
            if (x_row_end - x_row > 2)
            {
                sh_result[threadIdx.x + 2 * THREADS_X] = acc2;
                __syncthreads();
                if (threadIdx.x % 2 == 0)
                    atomicAdd((half2 *)out_.item_ptr(x_row + 2, w_column), HALF2(sh_result[2 * THREADS_X + threadIdx.x]));
                if (x_row_end - x_row > 2)
                {
                    sh_result[threadIdx.x + 3 * THREADS_X] = acc3;
                    __syncthreads();
                    if (threadIdx.x % 2 == 0)
                        atomicAdd((half2 *)out_.item_ptr(x_row + 3, w_column), HALF2(sh_result[3 * THREADS_X + threadIdx.x]));
                }
            }
        }
    }
#endif
}

fp_q4_matmul_kernel q4_matmul_kernel_pick(ExLlamaTuning *tuningParams, int block_size_z, int groupsize, uint32_t *x_map)
{
// <bool use_half2, bool use_groupsize, bool use_x_map>
#if defined(USE_ROCM)
    return q4_matmul_kernel<false, true, false>;
#endif
    if (tuningParams->matmul_no_half2)
    {
        if (block_size_z % groupsize == 0)
        {
            if (x_map)
                return q4_matmul_kernel<false, true, true>;
            else
                return q4_matmul_kernel<false, true, false>;
        }
        else
        {
            if (x_map)
                return q4_matmul_kernel<false, false, true>;
            else
                return q4_matmul_kernel<false, false, false>;
        }
    }
    else
    {
        if (block_size_z % groupsize == 0)
        {
            if (x_map)
                return q4_matmul_kernel<true, true, true>;
            else
                return q4_matmul_kernel<true, true, false>;
        }
        else
        {
            if (x_map)
                return q4_matmul_kernel<true, false, true>;
            else
                return q4_matmul_kernel<true, false, false>;
        }
    }
};

// Compute y = x @ w

void q4_matmul_cuda(
    ExLlamaTuning *tuningParams,
    const half *x,
    const int x_height,
    const Q4Matrix *w,
    half *out,
    bool no_zero,
    hipStream_t alt_stream)
{
    int height = x_height;
    int dim = w->height;
    int width = w->width;

    hipSetDevice(w->device);

    uint32_t *x_map = w->cuda_x_map;
    const half *x_mapped = x;
    if (x_map && !tuningParams->matmul_fused_remap && !alt_stream)
    {
        CudaBuffers *buffers = get_buffers(w->device);
        column_remap_cuda(x, buffers->temp_state, x_height, dim, w->cuda_x_map);
        x_mapped = buffers->temp_state;
        x_map = NULL;
    }

    int block_size_z;
    if (w->width == 4096)
        block_size_z = 384; // 7B
    else if (w->width == 11008)
        block_size_z = 256;
    else if (w->width == 5120)
        block_size_z = 384; // 13B
    else if (w->width == 13824)
        block_size_z = 256;
    else if (w->width == 6656)
        block_size_z = 256; // 33B
    else if (w->width == 17920)
        block_size_z = 128;
    else
        block_size_z = 256;
    int block_size_y = 4;
    // if (!no_zero) hipMemsetAsync(out, 0, x_height * w->width * sizeof(half));

    dim3 threads(THREADS_X, THREADS_Y, 1);
#ifndef WMMA_KERNEL
    dim3 blocks(
        (width + threads.x - 1) / threads.x,
        (height + threads.y - 1) / threads.y,
        (dim + block_size_z - 1) / block_size_z);
#else
    dim3 blocks(
        (width + threads.x - 1) / threads.x,
        (height + block_size_y - 1) / block_size_y,
        (dim + block_size_z - 1) / block_size_z);
#endif

    fp_q4_matmul_kernel kernel = q4_matmul_kernel_pick(tuningParams, block_size_z, w->groupsize, x_map);

#if defined(USE_SMEM)

    int shared_mem = (block_size_z % w->groupsize == 0 ? w->groupsize : GROUP_STEP) * sizeof(half);

#else

    int shared_mem = 0;

#endif

    kernel<<<blocks, threads, shared_mem, alt_stream>>>(x_mapped, 
                                                        w->cuda_qweight, 
                                                        out, 
                                                        w->cuda_scales, 
                                                        w->cuda_qzeros, 
                                                        height, 
                                                        dim, 
                                                        width, 
                                                        w->groupsize,
#ifdef WMMA_KERNEL
                                                        block_size_y,
#endif 
                                                        block_size_z, 
                                                        x_map, 
                                                        no_zero);
}

void q4_matmul_recons_cuda(
    ExLlamaTuning *tuningParams,
    const half *x,
    const int x_height,
    Q4Matrix *w,
    half *out,
    const hipblasHandle_t handle,
    bool no_zero)
{
    int height = x_height;
    int dim = w->height;
    int width = w->width;

    hipSetDevice(w->device);
    CudaBuffers *buffers = get_buffers(w->device);

    const half *x_mapped = x;
    if (w->cuda_x_map)
    {
        TORCH_CHECK(buffers->temp_state_size >= x_height * dim, "temp_state buffer is too small");
        column_remap_cuda(x, buffers->temp_state, x_height, dim, w->cuda_x_map);
        x_mapped = buffers->temp_state;
    }

    w->reconstruct(buffers->temp_dq);

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 700

    const float alpha = 1.0f;
    const float beta = no_zero ? 1.0f : 0.0f;
    cublasSgemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, width, height, dim, &alpha, buffers->temp_dq, HIP_R_16F, width,
                  x_mapped, HIP_R_16F, dim, &beta, out, HIP_R_16F, width);

#else

    const half alpha = __float2half(1.0f);
    const half beta = no_zero ? __float2half(1.0f) : __float2half(0.0f);
    hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, width, height, dim, &alpha, buffers->temp_dq, width, x_mapped, dim, &beta, out, width);

#endif
}